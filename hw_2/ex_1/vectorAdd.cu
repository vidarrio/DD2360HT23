

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

#define DataType double

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        //  Handle error
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here

  int id = blockDim.x * blockIdx.x + threadIdx.x;

  if (id < len) {
    out[id] = in1[id] + in2[id];
  }
}

int main(int argc, char **argv) {
  
  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  //@@ Insert code below to read in inputLength from args
  if (argc != 2) {
    printf("Usage: vecAdd inputLength\n");
    exit(1);
  }

  inputLength = atoi(argv[1]);
  
  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  hostInput1 = (DataType *) malloc(inputLength * sizeof(DataType));
  hostInput2 = (DataType *) malloc(inputLength * sizeof(DataType));
  hostOutput = (DataType *) malloc(inputLength * sizeof(DataType));
  
  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  srand(time(NULL));

  for (int i = 0; i < inputLength; i++) {
    hostInput1[i] = (DataType) rand() / RAND_MAX;
    hostInput2[i] = (DataType) rand() / RAND_MAX;
  }

  resultRef = (DataType *) malloc(inputLength * sizeof(DataType));

  for (int i = 0; i < inputLength; i++) {
    resultRef[i] = hostInput1[i] + hostInput2[i];
  }

  //@@ start cuda profiler
  hipProfilerStart();

  //@@ start cpu timer
  double cpu_start = get_wall_time();

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, inputLength * sizeof(DataType));
  hipMalloc(&deviceInput2, inputLength * sizeof(DataType));
  hipMalloc(&deviceOutput, inputLength * sizeof(DataType));

  //@@ Insert code to below to Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(DataType), hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(DataType), hipMemcpyHostToDevice);

  //@@ Initialize the 1D grid and block dimensions here

  // 1024 is the maximum number of threads per block for compute capability 6.1
  int threadsPerBlock = 1024;

  // round up to the nearest integer so we don't have a partial block
  int blocksPerGrid = (inputLength + threadsPerBlock - 1) / threadsPerBlock;

  std::cout << "Blocks per grid: " << blocksPerGrid << std::endl;
  std::cout << "Threads per block: " << threadsPerBlock << std::endl;

  dim3 dimGrid(blocksPerGrid, 1, 1);
  dim3 dimBlock(threadsPerBlock, 1, 1);

  //@@ Launch the GPU Kernel here
  vecAdd<<<dimGrid, dimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(DataType), hipMemcpyDeviceToHost);

  //@@ stop cpu timer
  double cpu_end = get_wall_time();

  //@@ stop cuda profiler
  hipProfilerStop();

  //@@ Insert code below to compare the output with the reference
  int errors = 0;
  for (int i = 0; i < inputLength; i++) {
    if (abs(hostOutput[i] - resultRef[i]) > 1e-5) {
      errors++;
    }
  }

  std::cout << "Error: " << errors << std::endl;

  //@@ Insert code below to print out the timing results
  std::cout << "Execution time: " << cpu_end - cpu_start << std::endl;

  //@@ Free the GPU memory here

  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(resultRef);
  
  return 0;
}
